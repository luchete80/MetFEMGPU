#include "Domain_d.cuh"
#include <iostream>

using namespace std;

namespace MetFEM{

	void __host__ Domain_d::SolveChungHulbert(){
	
	int N = getElemCount();
	threadsPerBlock = 256; //Or BlockSize
	//threadsPerBlock = 1; //Or BlockSize
	blocksPerGrid =				// Or gridsize
	(N + threadsPerBlock - 1) / threadsPerBlock;
	cout << "Blocks per grid"<<blocksPerGrid<<", Threads per block"<< threadsPerBlock<<endl;
	
	calcElemJAndDerivKernel<<<blocksPerGrid,threadsPerBlock >>>(this);
	hipDeviceSynchronize(); 
	}
	
};