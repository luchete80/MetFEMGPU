#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
#include <iostream>
#include <vector>

#include "tensor.cuh"
#include "Matrix.cuh"

using namespace std;

namespace MetFEM {

void Domain_d::SetDimension(const int &node_count, const int &elem_count){
  
  m_node_count = node_count;
  m_elem_count = elem_count;
  
	hipMalloc((void **)&x, node_count * sizeof (double3));
	hipMalloc((void **)&v, node_count * sizeof (double3));
	hipMalloc((void **)&a, node_count * sizeof (double3));
	
	report_gpu_mem_();

}

void Domain_d::AddBoxLength(double3 const & V, double3 const & L, const double &r){
	    // integer, intent(in):: tag
    // logical, intent(in) :: redint
    // !real(fp_kind), intent(in), allocatable :: V
    // real(fp_kind), dimension(1:3), intent(in)  :: V ! input
    // real(fp_kind), intent(in):: r, Lx, Ly, Lz, Density, h  
    double3 Xp;
    int p, nnodz;
    int nodxelem;
    int nel[3];
    m_dim = 3;
    if (L.z > 0.0) m_dim = 2;
    
    
    nel[0] = (int)(L.x/(2.0*r));
    nel[1] = (int)(L.y/(2.0*r));
    cout << "Nel x: "<<nel[0]<<", y "<<nel[1]<<endl;
    if (m_dim == 2){
      nel[2] = 1;
      nodxelem = 4;
    } else {
      nel[2] = (int)(L.z/(2.0*r));
      nodxelem = 8;
    }
    

    Xp.z = V.z ;
    

    // write (*,*) "Creating Mesh ...", "Elements ", neL.y, ", ",neL.z
  int nc = (nel[0] +1) * (nel[1]+1) * (nel[2]+1);
  int ne = nel[0]*nel[1]*nel[2];
  //thisAllocateNodes((nel[0] +1) * (nel[1]+1) * (nel[2]+1));
    // print *, "Element count in XYZ: ", nel(:)
    // write (*,*) "Box Node count ", node_count

	this->SetDimension(nc,ne);	 //AFTER CREATING DOMAIN
  cout << "Mesh generated. Node count: " << nc<<". Element count: "<<ne<<endl;
  //SPH::Domain	dom;
	//double3 *x =  (double3 *)malloc(dom.Particles.size());
	double3 *x_H =  new double3 [m_node_count];


	//int size = dom.Particles.size() * sizeof(double3);
	cout << "Copying to device..."<<endl;
    
    cout << "Box Particle Count is " << m_node_count <<endl;
    p = 0;
    for (int j = 0; j < (nel[1] +1);j++) {
      Xp.y = V.y;
      for (int j = 0; j < (nel[1] +1);j++){
        Xp.x = V.x;
        for (int i = 0; i < (nel[0] +1);i++){
					//m_node.push_back(new Node(Xp));
					x_H[p] = Xp;
          //nod%x(p,:) = Xp(:);
          cout << "node " << p <<"X: "<<Xp.x<<"Y: "<<Xp.y<<"Z: "<<Xp.z<<endl;
          p++;
          Xp.x = Xp.x + 2.0 * r;
        }
        Xp.y = Xp.y + 2.0 * r;
      }// 
      Xp.z = Xp.z + 2 * r;

    //cout <<"m_node size"<<m_node.size()<<endl;
    } 
		hipMemcpy(this->x, x_H, m_node_count, hipMemcpyHostToDevice);    

    // !! ALLOCATE ELEMENTS
    // !! DIMENSION = 2
    int gp = 1;
    if (m_dim == 2) {
      // if (redint .eqv. .False.) then
        // gp = 4
      // end if 
      //call AllocateElements(neL.y * neL.z,gp) !!!!REDUCED INTEGRATION
    } else {
      // if (redint .eqv. .False.) then
        // gp = 8
      // end if 
      // call AllocateElements(neL.y * neL.z*nel(3),gp) 
    }

		unsigned int *elnod_h = new unsigned int [m_elem_count * nodxelem]; //Flattened
    
		int ex, ey, ez;
		std::vector <int> n;
    if (m_dim == 2) {
			n.resize(4);
      int ei = 0;
      for (int ey = 0; ey < nel[1];ey++){
        for (int ex = 0; ex < nel[0];ex++){
        int iv[4];
        elnod_h[ei] = (nel[0]+1)*ey + ex;        iv[ei+1] = (nel[0]+1)*ey + ex+1;
        iv[2] = (nel[0]+1)*(ey+1) + ex+1;        iv[3] = (nel[0]+1)*(ey+1) + ex;
        // cout << i[]
						// n[0]= m_node[iv[0]];
						// n[1]= m_node[(nel[0]+1)*ey + ex+1];
						// n[2]= m_node[(nel[0]+1)*(ey+1)+ex+1];
						// n[3]= m_node[(nel[0]+1)*(ey+1)+ex];
            cout << "Nel x : "<<nel[0]<<endl;
           cout << "nodes "<<endl;
           for (int i=0;i<4;i++)cout << iv[i]<<", ";
						 //m_element.push_back(new El4N2DPE(n));
																							// m_node[(nel[0]+1)*ey + ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex]
																							// );
              //elem%elnod(i,:)=[(neL.y+1)*ey + ex+1,(neL.y+1)*ey + ex+2,(neL.y+1)*(ey+1)+ex+2,(neL.y+1)*(ey+1)+ex+1]         
              //print *, "Element ", i , "Elnod", elem%elnod(i,:) 
					 }
      } 
    } else { //dim: 3
      int ei = 0;
      int nnodz = (nel[0]+1)*(nel[1]+1);
      for (int ez = 0; ez < nel[2];ez++)
      for (int ey = 0; ey < nel[1];ey++){
        for (int ex = 0; ex < nel[0];ex++){
          
          int iv[8];
          int nb1 = nnodz*ez + (nel[0]+1)*ey + ex;
          int nb2 = nnodz*ez + (nel[0]+1)*(ey+1) + ex;
          elnod_h[ei  ] = nb1;
          elnod_h[ei+1] = nb1+1;
          elnod_h[ei+2] = nb2+1;
          elnod_h[ei+3] = nb2;
          elnod_h[ei+4] = nb1 + nnodz*(ez+1);
          elnod_h[ei+5] = nb1 + nnodz*(ez+1) + 1;
          elnod_h[ei+6] = nb2 + nnodz*(ez+1) + 1;
          elnod_h[ei+7] = nb2 + nnodz*(ez+1);

          // elem%elnod(i,:) = [ nnodz*ez + (nel(1)+1)*ey + ex+1,nnodz*ez + (nel(1)+1)*ey + ex+2, &
                              // nnodz*ez + (nel(1)+1)*(ey+1)+ex+2,nnodz*ez + (nel(1)+1)*(ey+1)+ex+1, &
                              // nnodz*(ez + 1) + (nel(1)+1)*ey + ex+1,nnodz*(ez + 1) + (nel(1)+1)*ey + ex+2, &
                              // nnodz*(ez + 1) + (nel(1)+1)*(ey+1)+ex+2,nnodz*(ez + 1)+ (nel(1)+1)*(ey+1)+ex+1];
        // cout << i[]
						// n[0]= m_node[iv[0]];
						// n[1]= m_node[(nel[0]+1)*ey + ex+1];
						// n[2]= m_node[(nel[0]+1)*(ey+1)+ex+1];
						// n[3]= m_node[(nel[0]+1)*(ey+1)+ex];
            cout << "Nel x : "<<nel[0]<<endl;
           cout << "nodes "<<endl;
           
           for (int i=0;i<4;i++)cout << iv[i]<<", ";
           ei += nodxelem;
						 //m_element.push_back(new El4N2DPE(n));
																							// m_node[(nel[0]+1)*ey + ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex]
																							// );
              //elem%elnod(i,:)=[(neL.y+1)*ey + ex+1,(neL.y+1)*ey + ex+2,(neL.y+1)*(ey+1)+ex+2,(neL.y+1)*(ey+1)+ex+1]         
              //print *, "Element ", i , "Elnod", elem%elnod(i,:) 
					 }
      } 

		}//if dim 
    
    // call AllocateDomain()
    // i = 1
    // do while ( i <= node_count)
      // nod%is_bcv(i,:) = .false.
      // i = i + 1
    // end do
  
    // ! nod%m(:)   = Density * Lx * Ly * Lz / node_count
    // ! nod%rho(:)   = Density
    // elem%rho_0(:,:) = Density
    // !print *, "Particle mass ", nod%m(2)
    
    // !nod%id(:) = tag
    
    // fext_glob (:,:) = 0.0d0
    
    // elem%e_length(:) = Lx !TODO: CHANGE!
    
    // tot_mass = Density * Lx * Ly * Lz
    // if (dim == 2) then !!!assuming plain strain
      // tot_mass = tot_mass / Lz
    // end if
    // print *, "Total Mass: ", tot_mass
    
    // call SearchNodelem
		
		delete [] elnod_h;
}

__device__ void Domain_d::calcDerivatives_FullInt () {
  
  int e = threadIdx.x + blockDim.x*blockIdx.x;
  if (e < m_elem_count) {
    
  // integer :: e
  // ! !rg=gauss[ig]
  // ! !sg=gauss[jg]
  // real(fp_kind), dimension(dim,nodxelem) :: dHrs !!! USED ONLY FOR SEVERAL GAUSS POINTS
  Matrix dHrs; /// IN ELEM_TYPE
  // real(fp_kind), dimension(nodxelem,dim) :: x2
  // real(fp_kind), dimension(dim,dim) :: test
  // real(fp_kind), dimension(dim, dim*nodxelem) :: temph
  
  // integer :: i,j,k, gp
  // real(fp_kind):: r   !!! USED ONLY FOR SEVERAL GAUSS POINTS
  // real(fp_kind), dimension(8,3):: gpc !!! gauss point coordinates, r,s,t
  
  // gp = 1
  // do e=1, elem_count
// ! #ifdef _PRINT_DEBUG_  
    // ! print *, "el ", e 
// ! #endif    
    // do i=1,nodxelem
        // !print *, "elnod " , elem%elnod(e,i)
        // x2(i,:)=nod%x(elem%elnod(e,i),:)
    // end do
    
    // if (elem%gausspc(e) .eq. 1) then      
    
      // if (dim .eq. 2) then 
        // !dHdrs [-1,1,1,-1;  -1.-1,1,1] x X2
        // !! J = [
        // !! dx/dr dy/dr
        // !! dx/ds dy/dx ]
        // !!! THIS IS TO AVOID MATMUL
        // ! print *, "nodes X ", x2(:,1)
        // ! print *, "nodes Y ", x2(:,2)
                
        // elem%jacob(e,gp,1,:) = -x2(1,:)+x2(2,:)+x2(3,:)-x2(4,:)
        // elem%jacob(e,gp,2,:) = -x2(1,:)-x2(2,:)+x2(3,:)+x2(4,:)
        // elem%jacob(e,gp,:,:) = 0.25*elem%jacob(e,gp,:,:)
        // else !!!DIM 3
          // !!!!! SETTING LIKE THIS AVOID MATMUL
          // elem%jacob(e,gp,1,:) = -x2(1,:)+x2(2,:)+x2(3,:)-x2(4,:)-x2(5,:)+x2(6,:)+x2(7,:)-x2(8,:)
          // elem%jacob(e,gp,2,:) = -x2(1,:)-x2(2,:)+x2(3,:)+x2(4,:)-x2(5,:)-x2(6,:)+x2(7,:)+x2(8,:)
          // elem%jacob(e,gp,3,:) = -x2(1,:)-x2(2,:)-x2(3,:)-x2(4,:)+x2(5,:)+x2(6,:)+x2(7,:)+x2(8,:)
          // !elem%jacob(e,gp,2,:) = [-x2(1,2),-x2(2,2), x2(3,2), x2(4,2),-x2(5,2),-x2(6,2), x2(7,2), x2(8,2)]
          // !elem%jacob(e,gp,3,:) = [-x2(1,3),-x2(2,3), x2(3,3), x2(4,3),-x2(5,3),-x2(6,3), x2(7,3), x2(8,3)]
          // ! dHrs(1,:)=[-1.0, 1.0, 1.0,-1.0,-1.0, 1.0, 1.0,-1.0] AND THIS IS dHrs*x2
          // ! dHrs(2,:)=[-1.0,-1.0, 1.0, 1.0,-1.0,-1.0, 1.0, 1.0]       
          // ! dHrs(3,:)=[-1.0,-1.0,-1.0,-1.0, 1.0, 1.0, 1.0, 1.0]  
          // ! elem%jacob(e,gp,1,:) = matmul(dHrs,x2)
          // elem%jacob(e,gp,:,:) = 0.125*elem%jacob(e,gp,:,:)
      // end if  !!!!DIM
      // elem%detJ(e,gp) = det(elem%jacob(e,gp,:,:))
    // else !!!!! GP > 1
      // r = 1.0/sqrt(3.0);
      // gpc(1,:)=[-r,-r,-r];   gpc(2,:)=[ r,-r,-r];      gpc(3,:)=[-r, r,-r];      gpc(4,:)=[ r, r,-r]; !These are the 4 points for 2D full elem
      // gpc(5,:)=[-r,-r, r];   gpc(6,:)=[ r,-r, r];      gpc(7,:)=[-r, r, r];      gpc(8,:)=[ r, r, r];
    
      if (m_dim == 3) {
        for (int gp=0;gp<gp_count;gp++){

          // dHrs(1,:)=[-1.0*(1-gpc(gp,2))*(1.0-gpc(gp,3)),     (1-gpc(gp,2))*(1.0-gpc(gp,3))&
                    // ,     (1+gpc(gp,2))*(1.0-gpc(gp,3)),-1.0*(1+gpc(gp,2))*(1.0-gpc(gp,3))&
                    // ,-1.0*(1-gpc(gp,2))*(1.0+gpc(gp,3)),     (1-gpc(gp,2))*(1.0+gpc(gp,3))&
                    // ,     (1+gpc(gp,2))*(1.0+gpc(gp,3)),-1.0*(1+gpc(gp,2))*(1.0+gpc(gp,3))]
          // dHrs(2,:)=[-1.0*(1-gpc(gp,1))*(1.0-gpc(gp,3)),-1.0*(1+gpc(gp,1))*(1.0-gpc(gp,3))&
                         // ,(1+gpc(gp,1))*(1.0-gpc(gp,3)),     (1-gpc(gp,1))*(1.0-gpc(gp,3))&
                    // ,-1.0*(1-gpc(gp,1))*(1.0+gpc(gp,3)),-1.0*(1+gpc(gp,1))*(1.0+gpc(gp,3))&
                         // ,(1+gpc(gp,1))*(1.0+gpc(gp,3)),     (1-gpc(gp,1))*(1.0+gpc(gp,3))]
          // dHrs(3,:)=[-1.0*(1-gpc(gp,1))*(1.0-gpc(gp,2)),-1.0*(1+gpc(gp,1))*(1.0-gpc(gp,2))&
                    // ,-1.0*(1+gpc(gp,1))*(1.0+gpc(gp,2)),-1.0*(1-gpc(gp,1))*(1.0+gpc(gp,2))&
                    // ,     (1-gpc(gp,1))*(1.0-gpc(gp,2)),     (1+gpc(gp,1))*(1.0-gpc(gp,2))&
                    // ,     (1+gpc(gp,1))*(1.0+gpc(gp,2)),     (1-gpc(gp,1))*(1.0+gpc(gp,2))]                     
          
          // elem%dHrs(e,gp,:,:) =  dHrs(:,:)         
          // !dHrs(2,:)=[(1+r(i)), (1-r(i)),-(1-r(i)),-(1+r(i))]         
          // !dHrs(3,:)=[(1+r(i)), (1-r(i)),-(1-r(i)),-(1+r(i))] 
          // !print *, "dhrs", dHrs 
          // !print *, "x2", x2 
          // elem%jacob(e,gp,:,:) = 0.125*matmul(dHrs,x2)
// ! #if defined _PRINT_DEBUG_
          // ! print *, "jacob ", elem%jacob(e,gp,:,:)
// ! #endif          
          // elem%detJ(e,gp) = det(elem%jacob(e,gp,:,:))
          // !print *, "detJ ", elem%detJ(e,gp)
        }
      } else { //!dim =2
        // do gp = 1,4
          // dHrs(1,:)=[-1.0*(1-gpc(gp,2)),     (1-gpc(gp,2))&
                    // ,     (1+gpc(gp,2)),-1.0*(1+gpc(gp,2))]
          // dHrs(2,:)=[-1.0*(1-gpc(gp,1)),-1.0*(1+gpc(gp,1))&
                         // ,(1+gpc(gp,1)),     (1-gpc(gp,1))]                
          
          // elem%dHrs(e,gp,:,:) =  dHrs(:,:)         
          // !dHrs(2,:)=[(1+r(i)), (1-r(i)),-(1-r(i)),-(1+r(i))]         
          // !dHrs(3,:)=[(1+r(i)), (1-r(i)),-(1-r(i)),-(1+r(i))] 
          // !print *, "dhrs", dHrs 
          // !print *, "x2", x2 
          // elem%jacob(e,gp,:,:) = 0.25*matmul(dHrs,x2)
// ! #if defined _PRINT_DEBUG_
          // !print *, "jacob ", elem%jacob(e,gp,:,:)
// ! #endif          
          // elem%detJ(e,gp) = det(elem%jacob(e,gp,:,:))
          // !print *, "detJ ", elem%detJ(e,gp)
        // end do !gp      
        
      }
    // end if !!gp ==1
// ! #if defined _PRINT_DEBUG_
    // !print *, "jacob ", elem%jacob(e,gp,:,:)
// ! #endif    
  // end do !element
  }
}


};
	