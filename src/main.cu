#include "hip/hip_runtime.h"
#include "cuda/Domain_d.cuh"

using namespace MetFEM;

int main(){

	Domain_d dom;
  
	double3 V = make_double3(0.0,0.0,0.0);
	double3 L = make_double3(0.1,0.1,0.1);
	double r = 0.05;
	
	dom.AddBoxLength(V,L,r);
	
}