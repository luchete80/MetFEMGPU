#include "hip/hip_runtime.h"
#include "cuda/Domain_d.cuh"

#include <iostream>
#include "cuda/cudautils.cuh"

using namespace MetFEM;

using namespace std;
void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    std::cout << "Free = " << free << " Total = " << total <<std::endl;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(){


	Domain_d *dom_d;

	report_gpu_mem();
	gpuErrchk(hipMallocManaged(&dom_d, sizeof(MetFEM::Domain_d)) );
	report_gpu_mem();
		
	double3 V = make_double3(0.0,0.0,0.0);
	double3 L = make_double3(0.1,0.1,0.1);  
	double r = 0.05;
	
	dom_d->AddBoxLength(V,L,r);
  
  double *a;
  hipFree(a);

  ////// MATERIAL  
  double E, nu;
  Material_ *mat_h = (Material_ *)malloc(dom_d->getElemCount() * sizeof(Material_ *)); 
  Elastic_ el(E,nu);
  // cout << "Mat type  "<<mattype<<endl;

  Material_ *material_h;
  double Ep, c[6];
  // MATERIAL
  //TODO: MATERIALS SHOULD BE A VECTOR
  
  string mattype = "Bilinear";
  if      (mattype == "Bilinear")    {
    Ep = E*c[0]/(E-c[0]);		                              //only constant is tangent modulus
    material_h  = new Bilinear(Ep);
    cout << "Material Constants, Et: "<<c[0]<<endl;
    // hipMalloc((void**)&dom_d->materials, 1 * sizeof(Bilinear )); //
    // hipMemcpy(dom_d->materials, material_h, 1 * sizeof(Bilinear), hipMemcpyHostToDevice);	
  } 
  // else if (mattype == "Hollomon")    {
    // // material_h  = new Hollomon(el,Fy,c[0],c[1]);
    // // cout << "Material Constants, K: "<<c[0]<<", n: "<<c[1]<<endl;
    // // hipMalloc((void**)&dom_d->materials, 1 * sizeof(Hollomon));
    
    // material_h  = new Material_(el);
    // material_h->InitHollomon(el,Fy,c[0],c[1]);
    // material_h->Material_model = HOLLOMON;
    // hipMalloc((void**)&dom_d->materials, 1 * sizeof(Material_));
    
    // //init_hollomon_mat_kernel<<<1,1>>>(dom_d); //CRASH
    // //hipMemcpy(dom_d->materials, material_h, 1 * sizeof(Hollomon*), hipMemcpyHostToDevice);	
    // hipMemcpy(dom_d->materials, material_h, 1 * sizeof(Material_), hipMemcpyHostToDevice);	 //OR sizeof(Hollomon)??? i.e. derived class
    
  
  // } else if (mattype == "JohnsonCook") {
    // //Order is 
                               // //A(sy0) ,B,  ,C,   m   ,n   ,eps_0,T_m, T_transition
   // //Material_ *material_h  = new JohnsonCook(el,Fy, c[0],c[1],c[3],c[2],c[6], c[4],c[5]); //First is hardening // A,B,C,m,n_,eps_0,T_m, T_t);	 //FIRST IS n_ than m
    
    // //Only 1 material to begin with
    // //hipMalloc((void**)&dom_d->materials, 1 * sizeof(JohnsonCook ));
    // //hipMemcpy(dom_d->materials, material_h, 1 * sizeof(JohnsonCook), hipMemcpyHostToDevice);	
    // cout << "Material Constants, B: "<<c[0]<<", C: "<<c[1]<<", n: "<<c[2]<<", m: "<<c[3]<<", T_m: "<<c[4]<<", T_t: "<<c[5]<<", eps_0: "<<c[6]<<endl;
  // } else                              printf("ERROR: Invalid material type.

	
	//SolverChungHulbert solver(&dom);
	cout << "Element Count "<<dom_d->getElemCount()<<endl;
	dom_d->SolveChungHulbert ();
	cout << "Program ended."<<endl;
	
	
}